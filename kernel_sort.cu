#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
//#include <hip/hip_runtime_api.h>

#include <time.h>
#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>

#define MaxStringLen 16                    //ʵ�ʿ��ó���-1
#define KeyNum 3
#define DataLine 7 
//#define DataLine 262144 //1024 * 256
//#define DataLine 98304  //1024 * 96 
//#define DataLine 1048576  //1024 * 1024 

const char path[100] = "TestData.txt";
const char path2[100] = "Data.txt";

struct KeyString
{
	char key[KeyNum][MaxStringLen];
};

bool __device__ __host__ operator < (KeyString l, KeyString r)
{
	for (int k = 0; k < KeyNum; k++)
	{
		for (long int i = 0; i < MaxStringLen; i++)
		{
			if (l.key[k][i] == '\0' && r.key[k][i] == '\0')    //into next key
			{
				if (k == KeyNum - 1)
					return false;
				break;
			}

			if (l.key[k][i] != '\0' && r.key[k][i] == '\0')
				return false;

			if (l.key[k][i] == '\0' && r.key[k][i] != '\0')
				return true;

			if (l.key[k][i] < r.key[k][i])
				return true;

			if (l.key[k][i] == r.key[k][i])
				continue;

			//if l >= r
			return false;                  
		}
	}
}

int main()
{
	const int key_num = KeyNum;
	const int dataline = DataLine;

	void GenerateRandomData(const int key_num, const int dataline);
	GenerateRandomData(key_num, dataline);

	//hipDeviceProp_t devprop;
	int deviceID = -1;
	if (hipSuccess == hipGetDevice(&deviceID))
	{
		printf("deviceID: %d\n", deviceID);
		hipDeviceProp_t devprop;
		hipGetDeviceProperties(&devprop, deviceID);
		long int total_memory;
		total_memory = sizeof(char) * KeyNum * MaxStringLen * DataLine + sizeof(long int) * DataLine;
		printf("%ld bytes memory used, %ld bytes total memory available.\n", total_memory, devprop.totalGlobalMem);
		if ( devprop.totalGlobalMem < total_memory )
		{
			printf("Error: insufficient amount of GPU memory.\n");
		}
	}

	thrust::host_vector<KeyString> key_vector(DataLine);
	thrust::host_vector<long int> value_vector(DataLine);

	FILE *fp;
	fp = fopen(path, "r+");                            //path�����ܲ��ԣ�path2���������ݲ���
	//fp = fopen(path2, "r+");
	for (long int i = 0; i < DataLine; i++)
	{

		for (int j = 0; j < key_num; j++)
		{
			fscanf(fp, "%s", key_vector[i].key[j]);
		}
		fscanf(fp, "%ld", &value_vector[i]);

		//printf("%s %s %s %ld\n", key_vector[i].key[0], key_vector[i].key[1], key_vector[i].key[2], value_vector[i]);
	}
	fclose(fp);

	printf("vector.size(): %ld\n", key_vector.size());
	printf("vector.capacity(): %ld\n", key_vector.capacity());

	hipEvent_t start_event, stop_event;

    hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	thrust::device_vector<KeyString> dkey_vector(DataLine);
	thrust::device_vector<long int> dvalue_vector(DataLine);

	hipEventRecord(start_event, 0);
	dkey_vector = key_vector;
	dvalue_vector = value_vector;

	printf("dvector.size(): %ld\n", dkey_vector.size());
	printf("dvector.capacity(): %ld\n", dkey_vector.capacity());

	thrust::sort_by_key(dkey_vector.begin(),dkey_vector.end(),dvalue_vector.begin());

	key_vector = dkey_vector;
	value_vector = dvalue_vector;

	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);

	for (long int i = 0; i < DataLine; i++)
	{
		for (int j = 0; j < key_num; j++)
		{
			printf("%s\t", key_vector[i].key[j]);
		}
		printf("%ld\n", value_vector[i]);

		if ( i > 100 )
		{
			break;
		}
	}

	float time = 0;
	hipEventElapsedTime(&time, start_event, stop_event);
	time /= 1.0e3f;
	printf("\nGPU ElapsedTime: %.5f s\n", time);

	//hipFree(dkey_vector);
	// Free vector memeory, ������ʱ�ռ����ͷ��ڴ�
	key_vector.clear();
	value_vector.clear();
	dkey_vector.clear();
	dvalue_vector.clear();
	thrust::host_vector<KeyString>(key_vector).swap(key_vector);
	thrust::host_vector<long int>(value_vector).swap(value_vector);
	thrust::device_vector<KeyString>(dkey_vector).swap(dkey_vector);
	thrust::device_vector<long int>(dvalue_vector).swap(dvalue_vector);

	return 0;
}

__host__ void GenerateRandomData(const int key_num, const int dataline)
{
	FILE *fp;
	fp = fopen(path2, "w+");
	thrust::default_random_engine rng(clock());
	thrust::uniform_int_distribution<unsigned int> u(0, 10000);

	for (long int i = 0; i < dataline; i++)
	{
		for (int j = 0; j < key_num; j++)
		{
			fprintf(fp, "%u\t", u(rng));
		}
		fprintf(fp, "%d\n", i + 1);
	}

	fclose(fp);
}
